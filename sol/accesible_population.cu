#include "hip/hip_runtime.h"
#include <fstream>
#include "accesible_population.h"
#include "helper.h"

__global__ void calculateAccesiblePopulation(float *lat, float *lon,
                                             int *pop, int *acc, 
                                             int N, int kmRange) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    float cs, dist;
    
    /**
     * Se calculeaza populatia accesibila pentru orasul i si 
     * orasele de la i + 1 la N. 
     * 
     * Prima data se caculeaza distranta dintre orase.
     * Daca distanta este mai mica de kmRange, la populatia
     * accesibila a orasului A se adauga populatia orasului
     * B si invers.
     * 
     * Daca distanta este mai mare de cat kmRande, for-ul inceteaza,
     * pentru ca coordonatele sunt sortate si pentru coordonatele ce 
     * ar urma distanta ar fi mai mare de cat kmRange.
     */

    for (int j = i + 1; j < N; j++) {
        cs = sin(lat[i]) * sin(lat[j]) * cos(lon[i] - lon[j]) 
            + cos(lat[i]) * cos(lat[j]);

        if (cs > 1) {
            cs = 1;
        } else if (cs < -1) {
            cs = -1;
        }

        dist =  6371.f * acos(cs);

        if (dist <= kmRange) {
            atomicAdd(&acc[i], pop[j]);
            atomicAdd(&acc[j], pop[i]);
        } else {
            break;
        }
    }
}


void processCityData(cityInfo *data, int N, int kmRange) {
    hipSetDevice(0);
    

    float *device_lat, *device_lon, *host_lat, *host_lon;
    int *device_pop, *device_acc, *host_pop, *host_acc;

    host_lat = (float*) malloc(sizeof(float) * 10000000);
    host_lon = (float*) malloc(sizeof(float) * 10000000);
    host_pop = (int*) malloc(sizeof(int) * 10000000);
    host_acc = (int*) malloc(sizeof(int) * 10000000);

    hipMallocManaged((void **)&device_lat, N * sizeof(float));
    hipMallocManaged((void **)&device_lon, N * sizeof(float));
    hipMallocManaged((void **)&device_pop, N * sizeof(int));
    hipMallocManaged((void **)&device_acc, N * sizeof(int));

    /**
     * Se stocheaza latitudine, longitudinea si populatia in vectori.
     * Acestia vor fi folositi pentru calcularea distantei si a populatiei
     * pentru fiecare doua orase.
     */
    for (int i = 0; i < N; i++) {
        host_lat[i] = data[i].lat;
        host_lon[i] = data[i].lon;
        host_pop[i] = data[i].pop;
        host_acc[i] = data[i].acc;
        
    }

    hipMemcpy(device_lat, host_lat, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_lon, host_lon, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(device_pop, host_pop, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_acc, host_acc, N * sizeof(int), hipMemcpyHostToDevice);

    const size_t block_size = 100;
    size_t blocks_no = N / block_size;

    if (N % block_size)
    ++blocks_no;

    // Se calculeaza populatia accesibila pentru orase.
    calculateAccesiblePopulation<<<blocks_no, block_size>>>(device_lat, device_lon,
                                    device_pop, device_acc, N, kmRange);
    hipDeviceSynchronize();

    hipMemcpy(host_acc, device_acc, N *sizeof(int), hipMemcpyDeviceToHost);
    
    // Se copiaza populatia accesibila pentru orase in structura.
    for (int i = 0; i < N; i++) {
        data[i].acc = host_acc[i];
    }

    free(host_lat);
    free(host_lon);
    free(host_pop);
    free(host_acc);

    hipFree(device_lat);
    hipFree(device_lon);
    hipFree(device_pop);
    hipFree(device_acc);
}

